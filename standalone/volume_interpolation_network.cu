#include "hip/hip_runtime.h"
#include "volume_interpolation_network.h"

#include <hip/hip_runtime.h>

#include <iostream>
#include <iomanip>
#include <fstream>
#include <vector>

#include "renderer_utils.cuh"
#include "cuda_utils.h"

#include <tinyformat.h>

#define CU_SAFE_CALL( err ) (err)
#ifdef _MSC_VER
#define release_assert(expression) (void)(                                                       \
            (!!(expression)) ||                                                              \
            (_wassert(_CRT_WIDE(#expression), _CRT_WIDE(__FILE__), (unsigned)(__LINE__)), 0) \
        )
#else
#define release_assert assert
#endif

const int renderer::VolumeInterpolationNetwork::OutputType2ChannelCount[3] = {
	1, 3, 4
};

static void writeString(std::ostream& o, const std::string& s)
{
	int l = s.length();
	o.write(reinterpret_cast<const char*>(&l), sizeof(int));
	o.write(s.data(), l);
}
static std::string loadString(std::istream& i)
{
	int l;
	i.read(reinterpret_cast<char*>(&l), sizeof(int));
	std::string s;
	s.resize(l);
	i.read((char*)s.data(), l);
	return s;
}

int renderer::InputParametrization::channelsOut() const
{
	if (numFourierFeatures > 0) {
		return 4 + (hasDirection ? 4 : 0) + 2 * numFourierFeatures;
	}
	else
	{
		return 3 + (hasDirection ? 3 : 0);
	}
}

bool renderer::InputParametrization::valid() const
{
	if (useDirectionInFourierFeatures && !hasDirection)
	{
		std::cerr << "useDirectionInFourierFeatures==true requires hasDirection==true, but hasDirection is false" << std::endl;
		return false;
	}
	if (fourierMatrix.size() % 3 != 0)
	{
		std::cerr << "Fourier matrix size not divisible by three" << std::endl;
		return false;
	}
	int fourierChannels = useDirectionInFourierFeatures ? 6 : 3;
	if (numFourierFeatures >= 0 && (numFourierFeatures != fourierMatrix.size() / fourierChannels))
	{
		std::cerr << "Fourier features specified, but number of rows in 'fourierMatrix' does not match 'numFourierFeatures" << std::endl;
		return false;
	}
	if ((numFourierFeatures % 2) != 0)
	{
		std::cerr << "The number of fourier features must be divisible by 2, but is " << numFourierFeatures << std::endl;
		return false;
	}
	return true;
}

const int renderer::InputParametrization::VERSION = 3;

renderer::InputParametrization_ptr renderer::InputParametrization::load(std::istream& in)
{
	auto p = std::make_shared<InputParametrization>();

	int version;
	in.read(reinterpret_cast<char*>(&version), sizeof(int));

	if (version == 1)
	{
		in.read(reinterpret_cast<char*>(&p->hasDirection), sizeof(bool));
		in.read(reinterpret_cast<char*>(&p->numFourierFeatures), sizeof(int));
		p->useDirectionInFourierFeatures = false;
		FourierMatrix_t m(p->numFourierFeatures * 3);
		if (p->numFourierFeatures > 0)
			in.read(reinterpret_cast<char*>(m.data()), sizeof(half) * p->numFourierFeatures * 3);
		p->fourierMatrix = m;
	}
	else if (version == 2)
	{
		in.read(reinterpret_cast<char*>(&p->hasDirection), sizeof(bool));
		in.read(reinterpret_cast<char*>(&p->numFourierFeatures), sizeof(int));
		in.read(reinterpret_cast<char*>(&p->useDirectionInFourierFeatures), sizeof(bool));
		int C = p->useDirectionInFourierFeatures ? 6 : 3;
		FourierMatrix_t m(p->numFourierFeatures * C);
		if (p->numFourierFeatures > 0)
			in.read(reinterpret_cast<char*>(m.data()), sizeof(half) * p->numFourierFeatures * C);
		p->fourierMatrix = m;
	}
	else if (version == 3)
	{
		in.read(reinterpret_cast<char*>(&p->hasTime), sizeof(bool));
		in.read(reinterpret_cast<char*>(&p->hasDirection), sizeof(bool));
		in.read(reinterpret_cast<char*>(&p->numFourierFeatures), sizeof(int));
		in.read(reinterpret_cast<char*>(&p->useDirectionInFourierFeatures), sizeof(bool));
		int C = p->useDirectionInFourierFeatures ? 6 : 3;
		FourierMatrix_t m(p->numFourierFeatures * C);
		if (p->numFourierFeatures > 0)
			in.read(reinterpret_cast<char*>(m.data()), sizeof(half) * p->numFourierFeatures * C);
		p->fourierMatrix = m;
	}
	else
		throw std::runtime_error("Unknown version for InputParametrization " + std::to_string(version));

	return p;
}

void renderer::InputParametrization::save(std::ostream& out) const
{
	out.write(reinterpret_cast<const char*>(&VERSION), sizeof(int));
	out.write(reinterpret_cast<const char*>(&hasTime), sizeof(bool));
	out.write(reinterpret_cast<const char*>(&hasDirection), sizeof(bool));
	out.write(reinterpret_cast<const char*>(&numFourierFeatures), sizeof(int));
	out.write(reinterpret_cast<const char*>(&useDirectionInFourierFeatures), sizeof(bool));
	int C = useDirectionInFourierFeatures ? 6 : 3;
	ASSERT_THROW(numFourierFeatures*C == fourierMatrix.size(), "numFourierFeatures*C == fourierMatrix.size()");
	if (numFourierFeatures > 0)
		out.write(reinterpret_cast<const char*>(fourierMatrix.data()), sizeof(half) * numFourierFeatures * C);
}

// void renderer::InputParametrization::setFourierMatrixFromTensor(const torch::Tensor& t, bool premultiplied)
// {
// 	CHECK_DIM(t, 2);
// 	int C = t.size(1);
// 	if (C == 3)
// 	{
// 		useDirectionInFourierFeatures = false;
// 	}
// 	else if (C == 6)
// 	{
// 		if (!hasDirection)
// 			throw std::runtime_error("hasDirection==false, but the fourier matrix has input channels for the direction");
// 		useDirectionInFourierFeatures = true;
// 	}
// 	else
// 	{
// 		throw std::runtime_error(tinyformat::format("Unrecognized number of input channels. Actual: %d, expected: 3 or 6", C));
// 	}
// 	at::Tensor t2 = t.cpu();
// 	CHECK_DTYPE(t2, c10::kFloat);
// 	const auto tAcc = t2.accessor<float, 2>();
// 	numFourierFeatures = t.size(0);
// 	fourierMatrix.resize(numFourierFeatures * C);
// 	for (int cout = 0; cout < numFourierFeatures; ++cout)
// 		for (int cin = 0; cin < C; ++cin)
// 			fourierMatrix[cout + numFourierFeatures*cin] = __float2half(
// 				(premultiplied ? 1 : 2 * M_PI) * tAcc[cout][cin]);
// }

void renderer::InputParametrization::disableFourierFeatures()
{
	numFourierFeatures = 0;
	useDirectionInFourierFeatures = false;
	fourierMatrix = FourierMatrix_t();
}

const std::string renderer::OutputParametrization::OutputModeNames[] = {
	"density",
	"density:direct",
	"rgbo",
	"rgbo:direct",
	"densitygrad",
	"densitygrad:direct",
	"densitygrad:cubic",
	"densitycurvature",
	"densitycurvature:direct"
};
const int renderer::OutputParametrization::OutputModeNumChannelsIn[] = {
	1, 1, 4, 4, 4, 4, 4, 6, 6
};
const int renderer::OutputParametrization::OutputModeNumChannelsOut[] = {
	1, 1, 4, 4, 1, 1, 1, 1, 1
};

renderer::OutputParametrization::OutputMode renderer::OutputParametrization::OutputModeFromString(const std::string& s)
{
	for (int i=0; i<_NUM_OUTPUT_MODES_; ++i)
	{
		if (OutputModeNames[i] == s) return OutputMode(i);
	}
	throw std::runtime_error("No output mode found matching string " + s);
}

int renderer::OutputParametrization::channelsIn() const
{
	return OutputModeNumChannelsIn[outputMode];
}

int renderer::OutputParametrization::channelsOut() const
{
	return OutputModeNumChannelsOut[outputMode];
}


const int renderer::OutputParametrization::VERSION = 1;

renderer::OutputParametrization_ptr renderer::OutputParametrization::load(std::istream& in)
{
	int version;
	in.read(reinterpret_cast<char*>(&version), sizeof(int));
	if (version != VERSION)
		throw std::runtime_error("Unknown version for OutputParametrization " + std::to_string(version));

	auto p = std::make_shared<OutputParametrization>();
	p->outputMode = OutputModeFromString(loadString(in));
	return p;
}

void renderer::OutputParametrization::save(std::ostream& out) const
{
	out.write(reinterpret_cast<const char*>(&VERSION), sizeof(int));
	writeString(out, OutputModeNames[outputMode]);
}

const std::string renderer::Layer::ActivationNames[] = {
	"ReLU",
	"Sine",
	"Snake",
	"SnakeAlt",
	"Sigmoid",
	"None"
};

renderer::Layer::Activation renderer::Layer::ActivationFromString(const std::string& s)
{
	for (int i = 0; i < _NUM_ACTIVATIONS_; ++i)
	{
		if (ActivationNames[i] == s) return Activation(i);
	}
	throw std::runtime_error("No output mode found matching string " + s);
}

bool renderer::Layer::valid(bool isOutputLayer) const
{
	return (weights.size() == channelsIn*channelsOut) &&
		(bias.size() == channelsOut) &&
		(isOutputLayer || (bias.size() % 4 == 0));
}

const int renderer::Layer::VERSION = 2;

renderer::Layer_ptr renderer::Layer::load(std::istream& in)
{
	int version;
	in.read(reinterpret_cast<char*>(&version), sizeof(int));
	if (version != 1 && version != 2)
		throw std::runtime_error("Unknown version for InputParametrization " + std::to_string(version));

	int rows, cols;
	in.read(reinterpret_cast<char*>(&rows), sizeof(int));
	in.read(reinterpret_cast<char*>(&cols), sizeof(int));

	weights_t fw(rows * cols);
	bias_t fb(rows);
	in.read(reinterpret_cast<char*>(fw.data()), sizeof(half) * rows * cols);
	in.read(reinterpret_cast<char*>(fb.data()), sizeof(half) * rows);

	Activation a = ActivationFromString(loadString(in));
	float activationParameter = 1;
	if (version == 2)
		in.read(reinterpret_cast<char*>(&activationParameter), sizeof(float));

	return std::make_shared<Layer>(cols, rows, fw, fb, a, activationParameter);
}

void renderer::Layer::save(std::ostream& out) const
{
	out.write(reinterpret_cast<const char*>(&VERSION), sizeof(int));

	int rows = channelsOut;
	int cols = channelsIn;
	out.write(reinterpret_cast<const char*>(&rows), sizeof(int));
	out.write(reinterpret_cast<const char*>(&cols), sizeof(int));

	out.write(reinterpret_cast<const char*>(weights.data()), sizeof(half) * rows * cols);
	out.write(reinterpret_cast<const char*>(bias.data()), sizeof(half) * rows);

	writeString(out, ActivationNames[activation]);
	out.write(reinterpret_cast<const char*>(&activationParameter), sizeof(float));
}

// renderer::LatentGrid::LatentGrid(const torch::Tensor& t_, Encoding encoding)
// {
// 	CHECK_DIM(t_, 5);
// 	CHECK_SIZE(t_, 0, 1);
// 	torch::Tensor t = t_.to(c10::kCPU, c10::kFloat);
// 
// 	gridChannels = t.size(1);
// 	gridSizeZ = t.size(2);
// 	gridSizeY = t.size(3);
// 	gridSizeX = t.size(4);
// 
// 	this->encoding = encoding;
// 	switch (encoding)
// 	{
// 	case FLOAT:
// 		initEncodingFloat(t);
// 		break;
// 	case BYTE_LINEAR:
// 		initEncodingByteLinear(t);
// 		break;
// 	case BYTE_GAUSSIAN:
// 		initEncodingByteGaussian(t);
// 		break;
// 	default:
// 		throw std::runtime_error("Unsupported encoding");
// 	}
// }

double renderer::LatentGrid::LastEncodingError = 0;

// void renderer::LatentGrid::initEncodingFloat(const torch::Tensor& t)
// {
// 	static_assert(sizeof(float) == 4, "What compiler did you smoke?");
// 	const auto tAcc = t.accessor<float, 5>();
// 	grid_t g(t.numel() * 4);
// 	float* data = reinterpret_cast<float*>(g.data());
// 	for (int c = 0; c < gridChannels; ++c) for (int z = 0; z < gridSizeZ; ++z) for (int y = 0; y < gridSizeY; ++y) for (int x = 0; x < gridSizeX; ++x)
// 	{
// 		int cHigh = c / 4;
// 		int cLow = c % 4;
// 		data[idx(cHigh, z, y, x, cLow)] = tAcc[0][c][z][y][x];
// 	}
// 	this->grid = g;
// 	LastEncodingError = 0;
// }

// void renderer::LatentGrid::initEncodingByteLinear(const torch::Tensor& t)
// {
// 	//compute minimal and maximal value
// 	std::vector<float> minValues(gridChannels);
// 	std::vector<float> maxValues(gridChannels);
// 	std::cout << "Add LatentGrid with linear encoding, min/max per channel:";
// 	for (int c = 0; c < gridChannels; ++c)
// 	{
// 		minValues[c] = torch::min(t.select(1, c)).item<float>();
// 		maxValues[c] = torch::max(t.select(1, c)).item<float>();
// 		std::cout << " " << std::setprecision(3) << minValues[c] << "/" << maxValues[c];
// 	}
// 	std::cout << std::endl;
// 	//transform and save
// 	gridOffsetOrMean.resize(gridChannels);
// 	gridScaleOrStd.resize(gridChannels);
// 	grid.resize(t.numel());
// 	const auto tAcc = t.accessor<float, 5>();
// 	double encodingError = 0;
// 	for (int c = 0; c < gridChannels; ++c)
// 	{
// 		gridOffsetOrMean[c] = minValues[c];
// 		gridScaleOrStd[c] = maxValues[c] - minValues[c];
// 		float invScale = 1.0f / std::max(1e-5f, maxValues[c] - minValues[c]);
// 		for (int z = 0; z < gridSizeZ; ++z) for (int y = 0; y < gridSizeY; ++y) for (int x = 0; x < gridSizeX; ++x)
// 		{
// 			float value = tAcc[0][c][z][y][x];
// 			float x01 = (value - minValues[c]) * invScale;
// 			int x255 = std::max(0, std::min(255, static_cast<int>(std::roundf(255 * x01))));
// 			int cHigh = c / 4;
// 			int cLow = c % 4;
// 			auto xFinal = static_cast<uint8_t>(x255);
// 			grid[idx(cHigh, z, y, x, cLow)] = xFinal;
// 			//reverse-conversion to validate error
// 			float valueReconstructed = gridOffsetOrMean[c] + xFinal / 255.0f * gridScaleOrStd[c];
// 			//std::cout << "v=" << std::setprecision(4) << value << " -> " << int(xFinal) << " -> " << valueReconstructed << std::endl;
// 			encodingError += std::abs(value - valueReconstructed);
// 		}
// 	}
// 	std::cout << "Average absolute encoding error: " << std::setprecision(6) <<
// 		(encodingError / t.numel()) << std::endl;
// 	LastEncodingError = encodingError / t.numel();
// }

// void renderer::LatentGrid::initEncodingByteGaussian(const torch::Tensor& t)
// {
// 	//compute mean and variance
// 	std::vector<float> meanValues(gridChannels);
// 	std::vector<float> stdValues(gridChannels);
// 	std::cout << "Add LatentGrid with gaussian encoding, mean/std per channel:";
// 	for (int c = 0; c < gridChannels; ++c)
// 	{
// 		auto [std, mean] = torch::std_mean(t.select(1, c));
// 		meanValues[c] = mean.item<float>();
// 		stdValues[c] = std.item<float>();
// 		std::cout << " " << std::setprecision(2) << meanValues[c] << "/" << 
// 			std::setprecision(3) << stdValues[c];
// 	}
// 	std::cout << std::endl;
// 	//transform and save
// 	gridOffsetOrMean.resize(gridChannels);
// 	gridScaleOrStd.resize(gridChannels);
// 	grid.resize(t.numel());
// 	const auto tAcc = t.accessor<float, 5>();
// 	double encodingError = 0;
// 	for (int c = 0; c < gridChannels; ++c)
// 	{
// 		gridOffsetOrMean[c] = meanValues[c];
// 		gridScaleOrStd[c] = stdValues[c];
// 		float invStd = 1.0f / std::max(1e-5f, stdValues[c]);
// 		for (int z = 0; z < gridSizeZ; ++z) for (int y = 0; y < gridSizeY; ++y) for (int x = 0; x < gridSizeX; ++x)
// 		{
// 			float vx = tAcc[0][c][z][y][x]; // vx \prop N(mean, std)
// 			float vxHat = (vx - meanValues[c]) * invStd; // vxHat \prop N(0,1)
// 			static constexpr float invSqrt2 = 0.7071067811865475244008443621048f; // 1/sqrt(2)
// 			float theta01 = 0.5f * (1 + std::erf(vxHat * invSqrt2));
// 			int theta255 = std::max(0, std::min(255, static_cast<int>(std::roundf(255 * theta01))));
// 			int cHigh = c / 4;
// 			int cLow = c % 4;
// 			auto xFinal = static_cast<uint8_t>(theta255);
// 			grid[idx(cHigh, z, y, x, cLow)] = xFinal;
// 			//reverse-conversion to validate error
// 			//constants copied from renderer_volume_tensorcores.cuh
// 			static constexpr float ENCODING_GAUSSIAN_EPSILON = 1e-4f;
// 			static constexpr float ENCODING_GAUSSIAN_2_MINUS_EPSILON = 2 - 1e-4f;
// 			static constexpr float ENCODING_GAUSSIAN_SQRT2 = 1.4142135623730950488016887242096980f;
// 			float tmp = ENCODING_GAUSSIAN_SQRT2 * myErfInv(ENCODING_GAUSSIAN_2_MINUS_EPSILON * (xFinal / 255.0f - 0.5f));
// 			float valueReconstructed = gridOffsetOrMean[c] + tmp * gridScaleOrStd[c];
// 			//std::cout << "v=" << std::setprecision(4) << vx << " -> " << int(xFinal) << " -> " << valueReconstructed << std::endl;
// 			encodingError += std::abs(vx - valueReconstructed);
// 		}
// 	}
// 	std::cout << "Average absolute encoding error: " << std::setprecision(6) <<
// 		(encodingError / t.numel()) << std::endl;
// 	LastEncodingError = encodingError / t.numel();
// }

bool renderer::LatentGrid::isValid() const
{
	if (gridChannels<=0 || gridSizeX<=0 || gridSizeY<=0 || gridSizeZ<=0)
	{
		std::cerr << "Error, LatentGrid: all dimensions must be positive" << std::endl;
		return false;
	}
	if (gridChannels%16 != 0)
	{
		std::cerr << "Error, LatentGrid: the number of channels must be divisible by 16" << std::endl;
		return false;
	}

	size_t entries = grid.size();
	size_t expected = bytesPerEntry() * gridChannels * gridSizeZ * gridSizeY * gridSizeX;
	if (entries != expected)
	{
		std::cerr << "Error, LatentGrid: illegal grid size" << std::endl;
		return false;
	}
	if (encoding != FLOAT)
	{
		if (gridOffsetOrMean.size() != gridChannels)
		{
			std::cerr << "Error, LatentGrid: gridOffsetOrMean must contain gridChannels entries" << std::endl;
			return false;
		}
		if (gridScaleOrStd.size() != gridChannels)
		{
			std::cerr << "Error, LatentGrid: gridScaleOrStd must contain gridChannels entries" << std::endl;
			return false;
		}
	}

	return true;
}

renderer::LatentGrid::GPUArray::GPUArray(int sizeX, int sizeY, int sizeZ, bool isFloat, const char* data)
	: array(nullptr), texture(0)
{
	//create array
	hipExtent extent = make_hipExtent(sizeX, sizeY, sizeZ);
	int bytesPerType = isFloat ? 4 : 1;
	int bitsPerType = 8 * bytesPerType;
	auto format = isFloat ? hipChannelFormatKindFloat : hipChannelFormatKindUnsigned;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
		bitsPerType, bitsPerType, bitsPerType, bitsPerType, format);
	CUDA_CHECK(hipMalloc3DArray(&array, &channelDesc, extent));

	hipMemcpy3DParms params = { 0 };
	params.srcPtr = make_hipPitchedPtr(const_cast<char*>(data),
		bytesPerType * sizeX * 4, sizeX, sizeY);
	params.dstArray = array;
	params.extent = extent;
	params.kind = hipMemcpyHostToDevice;
	CUDA_CHECK(hipMemcpy3D(&params));

	//create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(hipResourceDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = array;
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(hipTextureDesc));
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = isFloat ? hipReadModeElementType : hipReadModeNormalizedFloat;
	texDesc.normalizedCoords = 1;
	CUDA_CHECK(hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL));
}

renderer::LatentGrid::GPUArray::~GPUArray()
{
	if (texture) {
		CUDA_CHECK_NOEXCEPT(hipDestroyTextureObject(texture));
		texture = 0;
	}
	if (array)
	{
		CUDA_CHECK_NOEXCEPT(hipFreeArray(array));
		array = nullptr;
	}
}

void renderer::LatentGrid::clearGPUResources()
{
	gpuResources_.clear();
}

void renderer::LatentGrid::copyGridToGPU(bool skipIfAlreadyInitialized)
{
	if (skipIfAlreadyInitialized && !gpuResources_.empty()) return;
	ASSERT_THROW(gridChannels % 4 == 0, "gridChannels must be divisible by four");
	gpuResources_.resize(gridChannels / 4);
	size_t stride = bytesPerEntry() * 4 * gridSizeX * gridSizeY * gridSizeZ;
	for (int i=0; i<gridChannels/4; ++i)
	{
		gpuResources_[i] = std::make_shared<GPUArray>(
			gridSizeX, gridSizeY, gridSizeZ, encoding == FLOAT, grid.data() + (i * stride));
	}
}

hipTextureObject_t renderer::LatentGrid::getTexture(int index) const
{
	ASSERT_THROW(index >= 0 && index < gridChannels / 4, "index of of bounds");
	ASSERT_THROW(!gpuResources_.empty(), "GPU resources not yet created");
	return gpuResources_[index]->texture;
}

float4 renderer::LatentGrid::getOffsetOrMean(int index) const
{
	ASSERT_THROW(index >= 0 && index < gridChannels / 4, "index of of bounds");
	ASSERT_THROW(encoding != FLOAT, "encoding != FLOAT");
	index *= 4;
	return make_float4(
		gridOffsetOrMean[index], gridOffsetOrMean[index + 1],
		gridOffsetOrMean[index + 2], gridOffsetOrMean[index + 3]);
}

float4 renderer::LatentGrid::getScaleOrStd(int index) const
{
	ASSERT_THROW(index >= 0 && index < gridChannels / 4, "index of of bounds");
	ASSERT_THROW(encoding != FLOAT, "encoding != FLOAT");
	index *= 4;
	return make_float4(
		gridScaleOrStd[index], gridScaleOrStd[index + 1],
		gridScaleOrStd[index + 2], gridScaleOrStd[index + 3]);
}

const int renderer::LatentGrid::VERSION = 1;

std::shared_ptr<renderer::LatentGrid> renderer::LatentGrid::load(std::istream& in)
{
	int version;
	in.read(reinterpret_cast<char*>(&version), sizeof(int));
	if (version != VERSION)
		throw std::runtime_error("Unknown version for LatentGrid " + std::to_string(version));

	auto g = std::make_shared<LatentGrid>();
	in.read(reinterpret_cast<char*>(&g->encoding), sizeof(int));
	in.read(reinterpret_cast<char*>(&g->gridChannels), sizeof(int));
	in.read(reinterpret_cast<char*>(&g->gridSizeZ), sizeof(int));
	in.read(reinterpret_cast<char*>(&g->gridSizeY), sizeof(int));
	in.read(reinterpret_cast<char*>(&g->gridSizeX), sizeof(int));

	size_t entries = g->bytesPerEntry() * g->gridChannels * g->gridSizeZ * g->gridSizeY * g->gridSizeX;
	grid_t data(entries);
	in.read(data.data(), entries);
	g->grid = data;
	if (g->encoding != FLOAT)
	{
		g->gridOffsetOrMean.resize(g->gridChannels);
		g->gridScaleOrStd.resize(g->gridChannels);
		in.read(reinterpret_cast<char*>(g->gridOffsetOrMean.data()), sizeof(float) * g->gridChannels);
		in.read(reinterpret_cast<char*>(g->gridScaleOrStd.data()), sizeof(float) * g->gridChannels);
	}

	return g;
}

void renderer::LatentGrid::save(std::ostream& out) const
{
	if (!isValid())
		throw std::runtime_error("LatentGridTimeAndEnsemble is not valid, cannot save");
	out.write(reinterpret_cast<const char*>(&VERSION), sizeof(int));
	int e = static_cast<int>(encoding);
	out.write(reinterpret_cast<const char*>(&e), sizeof(int));
	out.write(reinterpret_cast<const char*>(&gridChannels), sizeof(int));
	out.write(reinterpret_cast<const char*>(&gridSizeZ), sizeof(int));
	out.write(reinterpret_cast<const char*>(&gridSizeY), sizeof(int));
	out.write(reinterpret_cast<const char*>(&gridSizeX), sizeof(int));
	size_t memory = bytesPerEntry() * gridChannels * gridSizeZ * gridSizeY * gridSizeX;
	const char* data = grid.data();
	out.write(data, memory);
	if (encoding != FLOAT)
	{
		out.write(reinterpret_cast<const char*>(gridOffsetOrMean.data()), sizeof(float)*gridChannels);
		out.write(reinterpret_cast<const char*>(gridScaleOrStd.data()), sizeof(float) * gridChannels);
	}
}

// double renderer::LatentGridTimeAndEnsemble::setTimeGridFromTorch(int index, const torch::Tensor& t,
// 															   LatentGrid::Encoding encoding)
// {
// 	ASSERT_THROW(index >= 0 && index < timeGrids.size(), "index out of bounds!");
// 	timeGrids[index] = std::make_shared<LatentGrid>(t, encoding);
// 	return LatentGrid::LastEncodingError;
// }

// double renderer::LatentGridTimeAndEnsemble::setEnsembleGridFromTorch(int index, const torch::Tensor& t,
// 	LatentGrid::Encoding encoding)
// {
// 	ASSERT_THROW(index >= 0 && index < ensembleGrids.size(), "index out of bounds!");
// 	ensembleGrids[index] = std::make_shared<LatentGrid>(t, encoding);
// 	return LatentGrid::LastEncodingError;
// }

bool renderer::LatentGridTimeAndEnsemble::isValid() const
{
	if (timeGrids.empty() && ensembleGrids.empty())
	{
		std::cerr << "Either time or ensemble grids must be specified!" << std::endl;
		return false;
	}

	//check common encoding
	LatentGrid::Encoding encoding = LatentGrid::FLOAT;
	bool encodingSet = false;
	for (auto g : timeGrids)
	{
		if (!g)
		{
			std::cerr << "One latent grid was null" << std::endl;
			return false;
		}
		if (!g->isValid())
			return false;
		if (encodingSet)
		{
			if (encoding != g->encoding)
			{
				std::cerr << "All latent grids must share the same encoding modes" << std::endl;
				return false;
			}
		}
		else
			encoding = g->encoding;
	}
	for (auto g : ensembleGrids)
	{
		if (!g)
		{
			std::cerr << "One latent grid was null" << std::endl;
			return false;
		}
		if (!g->isValid())
			return false;
		if (encodingSet)
		{
			if (encoding != g->encoding)
			{
				std::cerr << "All latent grids must share the same encoding modes" << std::endl;
				return false;
			}
		}
		else
			encoding = g->encoding;
	}

	//check common channel count
	if (!timeGrids.empty())
	{
		int c = timeGrids[0]->gridChannels;
		for (size_t i=1; i<timeGrids.size(); ++i)
		{
			if (timeGrids[i]->gridChannels != c)
			{
				std::cerr << "Time grid " << i << " uses a different channel count of " <<
					timeGrids[i]->gridChannels << " than previous grids with a channel count of " <<
					c << std::endl;
				return false;
			}
		}
	}
	if (!ensembleGrids.empty())
	{
		int c = ensembleGrids[0]->gridChannels;
		for (size_t i = 1; i < ensembleGrids.size(); ++i)
		{
			if (ensembleGrids[i]->gridChannels != c)
			{
				std::cerr << "Time grid " << i << " uses a different channel count of " <<
					ensembleGrids[i]->gridChannels << " than previous grids with a channel count of " <<
					c << std::endl;
				return false;
			}
		}
	}
	return true;
}

void renderer::LatentGridTimeAndEnsemble::clearGPUResources()
{
	for (auto g : timeGrids) g->clearGPUResources();
	for (auto g : ensembleGrids) g->clearGPUResources();
}

int renderer::LatentGridTimeAndEnsemble::getResolution() const
{
	assert(isValid());
	if (!timeGrids.empty()) return timeGrids[0]->gridSizeX;
	if (!ensembleGrids.empty()) return ensembleGrids[0]->gridSizeX;
	throw std::runtime_error("at least one grid must be active!");
}

renderer::LatentGrid::Encoding renderer::LatentGridTimeAndEnsemble::getCommonEncoding() const
{
	assert(isValid());
	if (!timeGrids.empty()) return timeGrids[0]->encoding;
	if (!ensembleGrids.empty()) return ensembleGrids[0]->encoding;
	throw std::runtime_error("at least one grid must be active!");
}

int renderer::LatentGridTimeAndEnsemble::getTimeChannels() const
{
	assert(isValid());
	if (timeGrids.empty()) return 0;
	return timeGrids[0]->gridChannels;
}

int renderer::LatentGridTimeAndEnsemble::getEnsembleChannels() const
{
	assert(isValid());
	if (ensembleGrids.empty()) return 0;
	return ensembleGrids[0]->gridChannels;
}

const int renderer::LatentGridTimeAndEnsemble::VERSION = 1;

renderer::LatentGridTimeAndEnsemble_ptr renderer::LatentGridTimeAndEnsemble::load(std::istream& in)
{
	int version;
	in.read(reinterpret_cast<char*>(&version), sizeof(int));
	if (version > VERSION)
		throw std::runtime_error("Unknown version for LatentGridTimeAndEnsemble " + std::to_string(version));

	auto g = std::make_shared<LatentGridTimeAndEnsemble>();
	in.read(reinterpret_cast<char*>(&g->timeMin), sizeof(int));
	in.read(reinterpret_cast<char*>(&g->timeNum), sizeof(int));
	in.read(reinterpret_cast<char*>(&g->timeStep), sizeof(int));
	in.read(reinterpret_cast<char*>(&g->ensembleMin), sizeof(int));
	in.read(reinterpret_cast<char*>(&g->ensembleNum), sizeof(int));
	std::cout << "Load " << g->timeNum << " time grids and " << g->ensembleNum << " ensemble grids" << std::endl;
	g->timeGrids.resize(g->timeNum);
	for (int i = 0; i < g->timeNum; ++i)
		g->timeGrids[i] = LatentGrid::load(in);
	g->ensembleGrids.resize(g->ensembleNum);
	for (int i = 0; i < g->ensembleNum; ++i)
		g->ensembleGrids[i] = LatentGrid::load(in);

	return g;
}

void renderer::LatentGridTimeAndEnsemble::save(std::ostream& out) const
{
	if (!isValid())
		throw std::runtime_error("LatentGridTimeAndEnsemble is not valid, cannot save");
	out.write(reinterpret_cast<const char*>(&VERSION), sizeof(int));
	out.write(reinterpret_cast<const char*>(&timeMin), sizeof(int));
	out.write(reinterpret_cast<const char*>(&timeNum), sizeof(int));
	out.write(reinterpret_cast<const char*>(&timeStep), sizeof(int));
	out.write(reinterpret_cast<const char*>(&ensembleMin), sizeof(int));
	out.write(reinterpret_cast<const char*>(&ensembleNum), sizeof(int));
	for (int i = 0; i < timeNum; ++i)
		timeGrids[i]->save(out);
	for (int i = 0; i < ensembleNum; ++i)
		ensembleGrids[i]->save(out);
}

renderer::SceneNetwork::SceneNetwork()
	: boxMin_{make_float3(-5.f)}
	, boxSize_{make_float3(1.f)}
	, input_{std::make_shared<InputParametrization>()}
	, output_{std::make_shared<OutputParametrization>()}
{
}

void renderer::SceneNetwork::addLayer(Layer_ptr layer)
{
	if (hidden_.empty() && input_->numFourierFeatures>0)
	{
		//first layer, modify to include skipped inputs
		Layer::weights_t wOld = layer->weights;
		Layer::weights_t wNew;
		int newChannelsIn = layer->channelsIn;
		if (!input_->hasTime) {
			if (input_->hasDirection)
			{
				newChannelsIn = layer->channelsIn + 2;
				wNew.resize((layer->channelsIn + 2) * layer->channelsOut);
				//position
				for (int cin = 0; cin < 3; cin++)
					for (int cout = 0; cout < layer->channelsOut; ++cout)
						wNew[cout * newChannelsIn + cin] = wOld[cout * layer->channelsIn + cin];
				//direction
				for (int cin = 0; cin < 3; cin++)
					for (int cout = 0; cout < layer->channelsOut; ++cout)
						wNew[cout * newChannelsIn + cin + 4] = wOld[cout * layer->channelsIn + cin + 3];
				//fourier + extra
				int numExtra = layer->channelsIn - 6;
				for (int cin = 0; cin < numExtra; cin++)
					for (int cout = 0; cout < layer->channelsOut; ++cout)
						wNew[cout * newChannelsIn + cin + 8] = wOld[cout * layer->channelsIn + cin + 6];
			}
			else
			{
				newChannelsIn = layer->channelsIn + 1;
				wNew.resize((layer->channelsIn + 1) * layer->channelsOut);
				//position
				for (int cin = 0; cin < 3; cin++)
					for (int cout = 0; cout < layer->channelsOut; ++cout)
						wNew[cout * newChannelsIn + cin] = wOld[cout * layer->channelsIn + cin];
				//fourier + extra
				int numExtra = layer->channelsIn - 3;
				for (int cin = 0; cin < numExtra; cin++)
					for (int cout = 0; cout < layer->channelsOut; ++cout)
						wNew[cout * newChannelsIn + cin + 4] = wOld[cout * layer->channelsIn + cin + 3];
			}
			hidden_.push_back(std::make_shared<Layer>(newChannelsIn, layer->channelsOut, wNew, layer->bias, layer->activation, layer->activationParameter));
		} else
		{
			//time adds an extra input after position -> only modify if we also have direction
			if (input_->hasDirection)
			{
				newChannelsIn = layer->channelsIn + 1;
				wNew.resize((layer->channelsIn + 1) * layer->channelsOut);
				//position + time
				for (int cin = 0; cin < 4; cin++)
					for (int cout = 0; cout < layer->channelsOut; ++cout)
						wNew[cout * newChannelsIn + cin] = wOld[cout * layer->channelsIn + cin];
				//direction
				for (int cin = 0; cin < 3; cin++)
					for (int cout = 0; cout < layer->channelsOut; ++cout)
						wNew[cout * newChannelsIn + cin + 4] = wOld[cout * layer->channelsIn + cin + 4];
				//fourier + extra
				int numExtra = layer->channelsIn - 6;
				for (int cin = 0; cin < numExtra; cin++)
					for (int cout = 0; cout < layer->channelsOut; ++cout)
						wNew[cout * newChannelsIn + cin + 8] = wOld[cout * layer->channelsIn + cin + 7];
				//add modified
				hidden_.push_back(std::make_shared<Layer>(newChannelsIn, layer->channelsOut, wNew, layer->bias, layer->activation, layer->activationParameter));
			} else
			{
				//no change needed
				hidden_.push_back(layer);
			}
		}
	}
	else if (layer->channelsIn < 16 || layer->channelsOut < 16)
	{
		//first layer (no fourier features) or last layer (to density/color)
		//transpose
		Layer::weights_t wOld = layer->weights;
		Layer::weights_t wNew;
		wNew.resize(wOld.size());
		for (int cout = 0; cout < layer->channelsOut; ++cout)
			for (int cin = 0; cin < layer->channelsIn; ++cin)
				wNew[cout + layer->channelsOut * cin] = wOld[cout * layer->channelsIn + cin];
		hidden_.push_back(std::make_shared<Layer>(layer->channelsIn, layer->channelsOut, wNew, layer->bias, layer->activation, layer->activationParameter));
	}
	else
	{
		//regular hidden layers, simply add
		hidden_.push_back(layer);
	}
}

// void renderer::SceneNetwork::addLayerFromTorch(const torch::Tensor& weights, const torch::Tensor& bias,
// 	Layer::Activation activation, float activationParameter)
// {
// 	CHECK_DIM(weights, 2);
// 	CHECK_DIM(bias, 1);
// 	CHECK_SIZE(weights, 0, bias.size(0));
// 
// 	at::Tensor weights2 = weights.cpu();
// 	at::Tensor bias2 = bias.cpu();
// 	const auto weightsAcc = weights2.accessor<float, 2>();
// 	const auto biasAcc = bias2.accessor<float, 1>();
// 
// 	int channelsIn = weights.size(1);
// 	int channelsOut = weights.size(0);
// 
// 	Layer::weights_t w(channelsIn * channelsOut);
// 	for (int cout = 0; cout < channelsOut; ++cout)
// 		for (int cin = 0; cin < channelsIn; ++cin)
// 			w[cout * channelsIn + cin] = __float2half(weightsAcc[cout][cin]);
// 
// 	Layer::bias_t b(bias.size(0));
// 	for (int col = 0; col < b.size(); ++col)
// 		b[col] = __float2half(biasAcc[col]);
// 
// 	addLayer(std::make_shared<Layer>(channelsIn, channelsOut, w, b, activation, activationParameter));
// }

void renderer::SceneNetwork::setTimeAndEnsemble(float time, int ensemble)
{
	if (!latentGrid())
	{
		std::cerr << "No latent grid specified, setTimeAndEnsemble has no effect" << std::endl;
		return;
	}
	//silently clamp
	currentTime_ = clamp(time, 
		static_cast<float>(latentGrid()->timeMin), 
		static_cast<float>(latentGrid()->timeMaxInclusive()));
	currentEnsemble_ = clamp(ensemble, latentGrid()->ensembleMin, latentGrid()->ensembleMaxInclusive());

	//clear cache, as time and ensemble are written to constant memory
	cacheConstantMemory_.clear();
}

bool renderer::SceneNetwork::valid() const
{
	if (!input()->valid())
	{
		std::cerr << "Input parametrization is invalid" << std::endl;
		return false;
	}
	if (latentGrid() && !latentGrid()->isValid())
	{
		std::cerr << "LatentGrid is invalid" << std::endl;
		return false;
	}
	if (latentGrid() && input()->numFourierFeatures==0)
	{
		std::cerr << "Currently, LatentGrid requires fourier features as well" << std::endl;
		return false;
	}

	int current_channel = input()->channelsOut();
	if (latentGrid()) current_channel += latentGrid()->getTotalChannels();
	for (size_t i = 0; i < hidden_.size(); ++i)
	{
		auto l = hidden_[i];
		if (l->channelsIn != current_channel)
		{
			std::cerr << "Invalid input channels at hidden layer " << i << ", expected " << current_channel
				<< ", got " << l->channelsIn << std::endl;
			return false;
		}
		if (!l->valid(i == hidden_.size()-1))
		{
			std::cerr << "Invalid hidden layer " << i <<
				", probably weights and bias don't match or are not a multiple of 4" << std::endl;
			return false;
		}
		current_channel = l->channelsOut;
	}
	if (current_channel != output()->channelsIn())
	{
		std::cerr <<
			"Output channels from the hidden layers don't match the expected channels for the output parametrization. "
			<< "Expected " << output()->channelsIn() << ", got " << current_channel << std::endl;
		return false;
	}
	return true;
}

int renderer::SceneNetwork::computeMaxWarps(bool onlySharedMemory, bool adjoint) const
{
	static const int maxShared = 48 * 1024;
	static const int maxConstant = 16 * 1024;
	static const int bytesPerEntry = sizeof(half);
	static const int warpSize = 32;
	static_assert(sizeof(half) == 2, "What compiler did you smoke?");

	int numShared = 0; //num entries
	int numConst = 0;
	//input
	if (input_->numFourierFeatures)
		numConst += input_->fourierMatrix.size();
	//hidden
	int lastChannels = input_->channelsOut();
	int maxChannels = lastChannels;
	for (const auto& l : hidden_)
	{
		//input layer (if not fourier features) and output layer are in constant memory
		if (l->channelsIn<16 || l->channelsOut<16)
		{
			numConst += l->weights.size() + l->bias.size();
		}
		else {
			numShared += l->weights.size() + l->bias.size();
		}
		lastChannels = l->channelsOut;
		maxChannels = max(maxChannels, lastChannels);
	}

	if (onlySharedMemory)
	{
		numShared += numConst;
		numConst = 0;
	}

	int entriesPerThread = maxChannels;
	if (adjoint)
	{
		entriesPerThread += (static_cast<int>(hidden_.size()) - 1) * maxChannels;
	}

	//scale with bytes per entry
	numShared *= bytesPerEntry;
	numConst *= bytesPerEntry;
	maxChannels *= bytesPerEntry;
	entriesPerThread *= bytesPerEntry;

	if (numConst > maxConstant)
		return -1; //constants out of bounds
	int numWarps = static_cast<int>(std::floor((maxShared - numShared) / static_cast<float>(entriesPerThread * warpSize)));
	if (numWarps <= 0)
		return -1; //shared memory out of bounds
	return numWarps;
}

int renderer::SceneNetwork::numParameters() const
{
	int numParameters = 0;
	//input
	if (input_->numFourierFeatures)
		numParameters += input_->fourierMatrix.size();
	//hidden
	for (const auto& l : hidden_)
	{
		numParameters += l->weights.size() + l->bias.size();
	}
	return numParameters;
}

const int renderer::SceneNetwork::VERSION = 2;

renderer::SceneNetwork_ptr renderer::SceneNetwork::load(std::istream& in)
{
	int version;
	in.read(reinterpret_cast<char*>(&version), sizeof(int));
	if (version > VERSION)
		throw std::runtime_error("Unknown version for InputParametrization " + std::to_string(version));

	SceneNetwork_ptr p = std::make_shared<SceneNetwork>();
	p->input_ = InputParametrization::load(in);
	p->output_ = OutputParametrization::load(in);
	int numLayers;
	in.read(reinterpret_cast<char*>(&numLayers), sizeof(int));
	for (int i = 0; i < numLayers; ++i)
		p->hidden_.push_back(Layer::load(in));
	in.read(reinterpret_cast<char*>(&p->boxMin_.x), sizeof(float3));
	in.read(reinterpret_cast<char*>(&p->boxSize_.x), sizeof(float3));
	if (version == 2)
	{
		//include latent grid
		char hasLatentGrid;
		in.read(&hasLatentGrid, 1);
		if (hasLatentGrid>0)
		{
			p->latentGrid_ = LatentGridTimeAndEnsemble::load(in);
		}
	}
	return p;
}

void renderer::SceneNetwork::save(std::ostream& out) const
{
	if (!valid())
		throw std::runtime_error("scene network is not valid, cannot save");
	out.write(reinterpret_cast<const char*>(&VERSION), sizeof(int));
	input_->save(out);
	output_->save(out);
	int numLayers = hidden_.size();
	out.write(reinterpret_cast<const char*>(&numLayers), sizeof(int));
	for (int i = 0; i < numLayers; ++i)
		hidden_[i]->save(out);
	out.write(reinterpret_cast<const char*>(&boxMin_.x), sizeof(float3));
	out.write(reinterpret_cast<const char*>(&boxSize_.x), sizeof(float3));
	char hasLatentGrid = latentGrid_ ? 1 : 0;
	out.write(&hasLatentGrid, 1);
	if (latentGrid_) latentGrid_->save(out);
}

void renderer::SceneNetwork::clearGPUResources()
{
	cacheConstantMemory_.clear();
	cacheDefines_.clear();
	if (latentGrid())
		latentGrid()->clearGPUResources();
}

std::string renderer::SceneNetwork::codeReturnType() const
{
	switch (output()->outputMode)
	{
	case OutputParametrization::DENSITY:
	case OutputParametrization::DENSITY_DIRECT:
		return "real_t";
	case OutputParametrization::RGBO:
	case OutputParametrization::RGBO_DIRECT:
		return "real4";
	case OutputParametrization::DENSITY_GRADIENT:
	case OutputParametrization::DENSITY_GRADIENT_DIRECT:
	case OutputParametrization::DENSITY_GRADIENT_CUBIC:
	case OutputParametrization::DENSITY_CURVATURE:
	case OutputParametrization::DENSITY_CURVATURE_DIRECT:
		return "real_t";
	default: throw std::runtime_error("Unknown output mode");
	}
}

bool renderer::SceneNetwork::supportsNormals() const
{
	return output_->isDensityGradient() || output_->isDensityCurvature();
}

std::string renderer::SceneNetwork::getDefines(
	const GlobalSettings& s, 
	int numWarps, bool firstAndLastInSharedMemory,
	GradientMode gradientMode) const
{
	// For time measures: keep the gradient mode (e.g. adjoint), even if no gradients
	// are needed in the end.
	//if (!s.volumeShouldProvideNormals) gradientMode = GradientMode::OFF_OR_DIRECT;

	if (output_->isColor()) gradientMode = GradientMode::OFF_OR_DIRECT;
	if (!cacheDefines_.empty() &&
		cacheNumWarps_ == numWarps &&
		cacheFirstAndLastInSharedMemory_ == firstAndLastInSharedMemory &&
		cacheGradientMode_ == gradientMode) {
		return cacheDefines_;
	}
	cacheNumWarps_ = numWarps;
	cacheFirstAndLastInSharedMemory_ = firstAndLastInSharedMemory;
	cacheGradientMode_ = gradientMode;

	bool hasVolumetricFeatures = latentGrid_ != nullptr;
	bool hasFourierFeatures = input_->numFourierFeatures > 0;

	int hiddenChannels = hasFourierFeatures ? hidden_[0]->channelsIn : hidden_[0]->channelsOut;
	std::cout << "hiddenChannels = " << hiddenChannels << std::endl;

	int latentGridChannelsDiv16 = 0;
	int latentGridEncoding = 0;
	if (hasVolumetricFeatures)
	{
		if (!latentGrid_->isValid()) throw std::runtime_error("Latent Grid invalid");
		latentGridChannelsDiv16 = latentGrid_->getTotalChannels() / 16;
		latentGridEncoding = static_cast<int>(latentGrid_->getCommonEncoding());
		hiddenChannels -= latentGridChannelsDiv16 * 16;
		std::cout << "Grid hiddenChannels = " << hiddenChannels << std::endl;
	}

	int numHiddenLayers = static_cast<int>(hidden_.size()) - 1; //last layer with scalar or color output is handled separately
	if (!hasFourierFeatures) numHiddenLayers--; //special first layer from position
	if (hasVolumetricFeatures) numHiddenLayers--; //first layer is explicitly handled
	for (int i = 1; i < hidden_.size(); ++i) {
		std::cout << "[" << i << "] inC = " << hidden_[i]->channelsIn << ", outC = " << hidden_[i]->channelsOut << ", hiddenC = " << hiddenChannels << std::endl;
		if (hidden_[i]->channelsIn != hiddenChannels) 
			throw std::runtime_error("Currently, all hidden layers must have the same size");
	}
	if (hiddenChannels % 16 != 0)
		throw std::runtime_error("Hidden channels must be a multiple of 16");
	if (numHiddenLayers<0)
		throw std::runtime_error("at least one hidden layer needed");

	auto activation = hidden_[0]->activation;
	for (int i = 1; i < hidden_.size() - 1; ++i)
		if (hidden_[i]->activation != activation)
			throw std::runtime_error("Currently, all hidden layers must have the same activation function");
	if (hidden_[hidden_.size() - 1]->activation != Layer::None)
		throw std::runtime_error("The last layer must have activation 'None'");

	int baseChannels = input_->hasDirection ? 8 : 4;
	if (hasFourierFeatures && input_->numFourierFeatures != ((hidden_[0]->channelsIn - baseChannels - latentGridChannelsDiv16*16) / 2))
		throw std::runtime_error(tinyformat::format(
			"If fourier features are defined, 2*num_fourier+%d==hidden[0].channelsIn() must hold. num_fourier=%d, channelsIn=%d",
			baseChannels, input_->numFourierFeatures, hidden_[0]->channelsIn));

	int directionMode = 0;
	if (input_->hasDirection)
		directionMode = input_->useDirectionInFourierFeatures ? 2 : 1;

	std::stringstream ss;
	ss << "#define BLOCK_SIZE " << (numWarps * 32) << "\n";
	ss << "#define NUM_HIDDEN_LAYERS " << numHiddenLayers << "\n";
	ss << "#define HIDDEN_CHANNELS_DIV16 " << (hiddenChannels / 16) << "\n";
	ss << "#define HAS_FOURIER_FEATURES " << (input_->numFourierFeatures > 0 ? 1 : 0) << "\n";
	ss << "#define NUM_FOURIER_FEATURES " << input_->numFourierFeatures << "\n";
	ss << "#define USE_DIRECTION " << directionMode << "\n";
	ss << "#define ACTIVATION " << Layer::ActivationNames[activation] << "\n";
	ss << "#define OUTPUT_MODE " << int(output_->outputMode) << "\n";
	ss << "#define FIRST_AND_LAST_IN_SHARED_MEMORY " << (firstAndLastInSharedMemory ? 1 : 0) << "\n";
	ss << "#define LATENT_GRID_CHANNELS_DIV16 " << latentGridChannelsDiv16 << "\n";
	ss << "#define LATENT_GRID_ENCODING " << latentGridEncoding << "\n";
	ss << "#define PASS_TIME_TO_NETWORK " << (input_->hasTime?1:0) << "\n";
	ss << "#define GRADIENT_MODE " << static_cast<int>(gradientMode) << "\n";
	//std::cout << "DEFINES:\n" << ss.str() << std::endl;
	cacheDefines_ = ss.str();
	return cacheDefines_;
}

std::vector<std::string> renderer::SceneNetwork::getIncludeFileNames(const GlobalSettings& s) const
{
	return { "renderer_volume_tensorcores.cuh" };
}

std::string renderer::SceneNetwork::getConstantDeclarationName(const GlobalSettings& s) const
{
	return "volumeInterpolationTensorcoresParameters";
}

std::string renderer::SceneNetwork::getPerThreadType(const GlobalSettings& s) const
{
	return "::kernel::VolumeInterpolationTensorcores";
}

void renderer::SceneNetwork::fillConstantMemory(
	const GlobalSettings& s, float fdStepsize, hipDeviceptr_t ptr,
	hipStream_t stream)
{
	if (!cacheConstantMemory_.empty() && fdStepsize==cacheFDStepsize_)
	{
		CU_SAFE_CALL(hipMemcpyHtoDAsync(ptr, cacheConstantMemory_.data(), cacheConstantMemory_.size(), stream));
		return;
	}
	cacheFDStepsize_ = fdStepsize;

	static std::vector<char> MEMORY(1024 * 1024);

	const bool hasFourierFeatures = input_->numFourierFeatures > 0;
	const bool hasDirection = input_->hasDirection;
	const bool hasColorOutput = output_->outputMode == OutputParametrization::RGBO || output_->outputMode == OutputParametrization::RGBO_DIRECT;
	
	size_t index = 0;
	const auto addWithPadding = [&](const void* mem, size_t len, int padding = 32)
	{
		//add padding
		index = kernel::roundUpPower2(index, padding);
		if (len > 0) {
			release_assert(index + len < MEMORY.size());
			memcpy(MEMORY.data() + index, mem, len);
			index += len;
		}
	};

	if (hasFourierFeatures)
	{
		addWithPadding(input_->fourierMatrix.data(), sizeof(half) * input_->fourierMatrix.size()); //cWeightsFourier
	}
	else
	{
		int C = hasDirection ? 6 : 3;
		release_assert(hidden_[0]->channelsIn == C);
		addWithPadding(hidden_[0]->weights.data(), sizeof(half) * C * hidden_[0]->channelsOut); //cWeightsFirst
		addWithPadding(hidden_[0]->bias.data(), sizeof(half) * hidden_[0]->channelsOut); //cBiasFirst
	}

#define DUMP_ARRAY_HALF(ax, count)	\
	do {printf(#ax ":"); for (int ii=0; ii<(count); ++ii) {printf(" %.2f", __half2float(ax[ii]));} printf("\n"); } while(0)
#define DUMP_ARRAY_INT(ax, count)	\
	do {printf(#ax ":"); for (int ii=0; ii<(count); ++ii) {printf(" %d", int(ax[ii]));} printf("\n"); } while(0)
#define DUMP_ARRAY_FLOAT4(ax, count)	\
	do {printf(#ax ":"); for (int ii=0; ii<(count); ++ii) {	\
		printf(" %.2f", ax[ii].x);	\
		printf(" %.2f", ax[ii].y);	\
		printf(" %.2f", ax[ii].z);	\
		printf(" %.2f", ax[ii].w);	\
	} printf("\n"); } while(0)

	bool hasLatentGrid = latentGrid() != nullptr;
	if (hasLatentGrid)
	{
		int gridTimeChannels = latentGrid()->getTimeChannels();
		int gridEnsembleChannels = latentGrid()->getEnsembleChannels();
		int gridTotalChannels = latentGrid()->getTotalChannels();
		int gridTotalChannelsDiv4 = gridTotalChannels / 4;
		bool hasOffsetScale = latentGrid()->getCommonEncoding() != LatentGrid::FLOAT;

		//memory to be stored in the constant buffer
		std::vector<hipTextureObject_t> cLatentGridA(gridTotalChannelsDiv4);
		std::vector<hipTextureObject_t> cLatentGridB(gridTotalChannelsDiv4);
		std::vector<float4> cLatentGridOffsetA(gridTotalChannelsDiv4);
		std::vector<float4> cLatentGridOffsetB(gridTotalChannelsDiv4);
		std::vector<float4> cLatentGridScaleA(gridTotalChannelsDiv4);
		std::vector<float4> cLatentGridScaleB(gridTotalChannelsDiv4);
		std::vector<float4> cLatentGridInterpolation(gridTotalChannelsDiv4);

		//time interpolation
		if (latentGrid()->hasTimeGrids()) {
			float time = latentGrid()->interpolateTime(currentTime_);
			int timeLow = std::min(static_cast<int>(time), latentGrid()->timeNum - 1);
			int timeHigh = std::min(timeLow + 1, latentGrid()->timeNum-1);
			LatentGrid_ptr gridLow = latentGrid()->getTimeGrid(timeLow);
			LatentGrid_ptr gridHigh = latentGrid()->getTimeGrid(timeHigh);
			gridLow->copyGridToGPU(true);
			gridHigh->copyGridToGPU(true);
			for (int i = 0; i < gridTimeChannels / 4; ++i)
			{
				cLatentGridA[i] = gridLow->getTexture(i);
				cLatentGridB[i] = gridHigh->getTexture(i);
				if (hasOffsetScale)
				{
					cLatentGridOffsetA[i] = gridLow->getOffsetOrMean(i);
					cLatentGridOffsetB[i] = gridHigh->getOffsetOrMean(i);
					cLatentGridScaleA[i] = gridLow->getScaleOrStd(i);
					cLatentGridScaleB[i] = gridHigh->getScaleOrStd(i);
				}
				cLatentGridInterpolation[i] = make_float4(time);
			}
		}

		//ensemble interpolation (currently: no interpolation)
		if (latentGrid()->hasEnsembleGrids()) {
			int ensemble = latentGrid()->interpolateEnsemble(currentEnsemble_);
			LatentGrid_ptr grid = latentGrid()->getEnsembleGrid(ensemble);
			grid->copyGridToGPU(true);
			int iOff = gridTimeChannels / 4;
			for (int i = 0; i < gridEnsembleChannels / 4; ++i)
			{
				cLatentGridA[i + iOff] = grid->getTexture(i);
				cLatentGridB[i + iOff] = grid->getTexture(i);
				if (hasOffsetScale)
				{
					cLatentGridOffsetA[i + iOff] = grid->getOffsetOrMean(i);
					cLatentGridOffsetB[i + iOff] = grid->getOffsetOrMean(i);
					cLatentGridScaleA[i + iOff] = grid->getScaleOrStd(i);
					cLatentGridScaleB[i + iOff] = grid->getScaleOrStd(i);
				}
				cLatentGridInterpolation[i + iOff] = make_float4(0);
			}
		}

		//copy to data
		addWithPadding(cLatentGridA.data(), cLatentGridA.size() * sizeof(hipTextureObject_t));
		addWithPadding(cLatentGridB.data(), cLatentGridA.size() * sizeof(hipTextureObject_t));
		if (hasOffsetScale)
		{
			addWithPadding(cLatentGridOffsetA.data(), cLatentGridOffsetA.size() * sizeof(float4));
			addWithPadding(cLatentGridOffsetB.data(), cLatentGridOffsetB.size() * sizeof(float4));
			addWithPadding(cLatentGridScaleA.data(), cLatentGridScaleA.size() * sizeof(float4));
			addWithPadding(cLatentGridScaleB.data(), cLatentGridScaleB.size() * sizeof(float4));
		}
		addWithPadding(cLatentGridInterpolation.data(), cLatentGridInterpolation.size() * sizeof(float4));

		//weights for the first layer
		auto layer = hidden_[0];
		int channelsIn = layer->channelsIn;
		int channelsOut = layer->channelsOut;
		addWithPadding(layer->weights.data(), sizeof(half) * channelsIn * channelsOut);
		addWithPadding(layer->bias.data(), sizeof(half) * channelsOut);

		//DUMP_ARRAY_HALF(layer->weights, channelsIn* channelsOut);
		//DUMP_ARRAY_HALF(layer->bias, channelsOut);
	}

	int startHidden = (hasFourierFeatures && !hasLatentGrid) ? 0 : 1;
	int numHidden = static_cast<int>(hidden_.size()) - startHidden - 1;
	if (numHidden < 0) throw std::runtime_error("Number of (real) hidden channels is negative. How did that happen?");
	int hiddenChannels = hasFourierFeatures ? hidden_[startHidden]->channelsIn : hidden_[startHidden]->channelsOut;
	std::vector<half> weightsHidden(max(1, numHidden * hiddenChannels * hiddenChannels));
	std::vector<half> biasHidden(max(1, numHidden * hiddenChannels));
	for (int i=startHidden; i<hidden_.size()-1; ++i)
	{
		memcpy(
			weightsHidden.data() + ((i - startHidden) * hiddenChannels * hiddenChannels),
			hidden_[i]->weights.data(),
			sizeof(half) * hiddenChannels * hiddenChannels);
		memcpy(
			biasHidden.data() + ((i - startHidden) * hiddenChannels),
			hidden_[i]->bias.data(),
			sizeof(half) * hiddenChannels);
	}
	addWithPadding(weightsHidden.data(), weightsHidden.size() * sizeof(half)); //cWeightsHidden
	addWithPadding(biasHidden.data(), biasHidden.size() * sizeof(half)); //cBiasHidden
	//DUMP_ARRAY_HALF(weightsHidden, weightsHidden.size());
	//DUMP_ARRAY_HALF(biasHidden, biasHidden.size());

	int lastHidden = static_cast<int>(hidden_.size()) - 1;
	addWithPadding(hidden_[lastHidden]->weights.data(), sizeof(half) * hidden_[lastHidden]->weights.size()); //cWeightsLast
	addWithPadding(hidden_[lastHidden]->bias.data(), sizeof(half) * hidden_[lastHidden]->bias.size()); //cBiasLiast

	addWithPadding(&boxMin_.x, sizeof(float3));
	addWithPadding(&boxSize_.x, sizeof(float3));

	float activationParameter = hidden_[0]->activationParameter;
	for (int i = 1; i < hidden_.size() - 1; ++i)
		if (hidden_[startHidden]->activationParameter != activationParameter)
			throw std::runtime_error("Extra parameter of the activation must be the same over all layers");
	addWithPadding(&activationParameter, sizeof(float));
	addWithPadding(&fdStepsize, sizeof(float)); //finiteDifferencesStepsize
	addWithPadding(&fdStepsize, sizeof(float)); //latentGridDifferencesStepSize

	addWithPadding(nullptr, 0); //pad whole struct
	//write out
	cacheConstantMemory_.resize(index);
	std::copy_n(MEMORY.begin(), index, cacheConstantMemory_.begin());
	CU_SAFE_CALL(hipMemcpyHtoDAsync(ptr, MEMORY.data(), index, stream));
}

renderer::VolumeInterpolationNetwork::VolumeInterpolationNetwork()
	: selectedNetwork_(0)
	, gradientMode_(SceneNetwork::GradientMode::OFF_OR_DIRECT)
	, finiteDifferencesStepsize_(1/256.f)
    , adjointLatentGridCentralDifferencesStepsizeScale_(4)
	, onlySharedMemory_(false)
{
}

void renderer::VolumeInterpolationNetwork::addNetwork(
	SceneNetwork_ptr network, const std::string& filename)
{
	int numWarpsSharedOnly = network->computeMaxWarps(true, false);
	int numWarpsMixed = network->computeMaxWarps(false, false);
	if (numWarpsSharedOnly < 0 && numWarpsMixed < 0)
		throw std::runtime_error("The network is too large!");
	networks_.push_back({
		network,
		numWarpsSharedOnly,
		numWarpsMixed,
		network->computeMaxWarps(true, true),
		network->computeMaxWarps(false, true),
		filename,
		filename.empty() ? "" : std::filesystem::path(filename).filename().string()
		});
	selectNetwork(networks_.size() - 1);
}

void renderer::VolumeInterpolationNetwork::selectNetwork(int index)
{
	selectedNetwork_ = index;
	auto net = networks_[index].network;
	//call the super-methods to not overwite the network settings
	setBoxMin(make_double3(net->boxMin()));
	setBoxMax(make_double3(net->boxSize()+net->boxMin()));
	if (net->latentGrid())
	{
		hasTimesteps_ = net->latentGrid()->hasTimeGrids();
		currentTimestep_ = hasTimesteps_
			? clamp(currentTimestep_, static_cast<float>(net->latentGrid()->timeMin), static_cast<float>(net->latentGrid()->timeMaxInclusive()))
			: 0;
		hasEnsembles_ = net->latentGrid()->hasEnsembleGrids();
		currentEnsemble_ = hasEnsembles_
			? clamp(currentEnsemble_, net->latentGrid()->ensembleMin, net->latentGrid()->ensembleMaxInclusive())
			: 0;
	} else
	{
		hasTimesteps_ = false;
		currentTimestep_ = 0;
		hasEnsembles_ = false;
		currentEnsemble_ = 0;
	}
}

void renderer::VolumeInterpolationNetwork::loadNetwork(const std::string& filename)
{
	std::ifstream in(filename, std::ofstream::binary);
	if (!in.is_open()) throw std::runtime_error("Unable to open the file");
	addNetwork(SceneNetwork::load(in), filename);
}

void renderer::VolumeInterpolationNetwork::setNetwork(SceneNetwork_ptr network)
{
	networks_.clear();
	addNetwork(network, "");
}

renderer::SceneNetwork_ptr renderer::VolumeInterpolationNetwork::currentNetwork() const
{
	if (networks_.empty() || selectedNetwork_ < 0)
		throw std::runtime_error("No network loaded");
	return networks_[selectedNetwork_].network;;
}

void renderer::VolumeInterpolationNetwork::setTimeAndEnsemble(float time, int ensemble)
{
	auto net = networks_[selectedNetwork_].network;
	net->setTimeAndEnsemble(time, ensemble);
}

void renderer::VolumeInterpolationNetwork::setBoxMin(const double3& box_min)
{
    // setBoxMin(box_min);
	boxMin_ = box_min;

	for (auto& net : networks_)
	{
		net.network->setBoxMin(make_float3(boxMin()));
		net.network->setBoxSize(make_float3(boxSize()));
		net.network->clearGPUResources();
	}
}

void renderer::VolumeInterpolationNetwork::setBoxMax(const double3& box_max)
{
    // setBoxMax(box_max);
	boxMax_ = box_max;

	for (auto& net : networks_)
	{
		net.network->setBoxMin(make_float3(boxMin()));
		net.network->setBoxSize(make_float3(boxSize()));
		net.network->clearGPUResources();
	}
}

std::string renderer::VolumeInterpolationNetwork::getName() const
{
	return "SRN";
}

// void renderer::VolumeInterpolationNetwork::loadNetworkDialog()
// {
// 	std::cout << "Open file dialog" << std::endl;
// 
// 	// open file dialog
// 	auto results = pfd::open_file(
// 		"Load scene network",
// 		"",
// 		{ "Scene Networks", "*.volnet" },
// 		false
// 	).result();
// 	if (results.empty())
// 		return;
// 	std::string fileNameStr = results[0];
// 
// 	//auto fileNamePath = std::filesystem::path(fileNameStr);
// 	//sceneNetworkDirectory_ = fileNamePath.string();
// 	//ImGui::MarkIniSettingsDirty();
// 	//ImGui::SaveIniSettingsToDisk(GImGui->IO.IniFilename);
// 
// 	//load the file
// 	std::cout << "Load " << fileNameStr << std::endl;
// 	loadNetwork(fileNameStr);
// }

// bool renderer::VolumeInterpolationNetwork::drawUI(UIStorage_t& storage)
// {
// 	bool changed = false;
// 
// 	std::vector<const char*> networkNames(networks_.size());
// 	for (int i = 0; i < networks_.size(); ++i)
// 		networkNames[i] = networks_[i].humanname.c_str();
// 	if (ImGui::ListBox("", &selectedNetwork_, networkNames.data(), networks_.size()))
// 	{
// 		selectNetwork(selectedNetwork_);
// 		changed = true;
// 	}
// 	ImGui::SameLine();
// 	ImGui::BeginGroup();
// 	if (ImGui::Button(ICON_FA_FOLDER_OPEN "##Network"))
// 	{
// 		loadNetworkDialog();
// 		changed = true;
// 	}
// 	if (ImGui::ButtonEx(ICON_FA_MINUS "##Network", ImVec2(0, 0),
// 		networks_.empty() ? ImGuiButtonFlags_Disabled : 0))
// 	{
// 		networks_.erase(networks_.begin() + selectedNetwork_);
// 		selectNetwork(std::max(0, selectedNetwork_ - 1));
// 		changed = true;
// 	}
// 	ImGui::EndGroup();
// 
// 	if (hasTimesteps_)
// 	{
// 		auto grid = networks_[selectedNetwork_].network->latentGrid();
// 		if (ImGui::SliderFloat("Time##Network", &currentTimestep_,
// 			static_cast<float>(grid->timeMin), static_cast<float>(grid->timeMaxInclusive())))
// 		{
// 			setTimeAndEnsemble(currentTimestep_, currentEnsemble_);
// 			changed = true;
// 		}
// 	}
// 	if (hasEnsembles_)
// 	{
// 		auto grid = networks_[selectedNetwork_].network->latentGrid();
// 		if (ImGui::SliderInt("Ensemble##Network", &currentEnsemble_,
// 			grid->ensembleMin, grid->ensembleMaxInclusive()))
// 		{
// 			setTimeAndEnsemble(currentTimestep_, currentEnsemble_);
// 			changed = true;
// 		}
// 	}
// 
// 	if (ImGui::Checkbox("Use only shared memory", &onlySharedMemory_))
// 	{
// 		changed = true;
// 	}
// 
// 	const auto currentGradientMode = magic_enum::enum_name(gradientMode_);
// 	if (ImGui::BeginCombo("Gradient Mode##Network", currentGradientMode.data()))
// 	{
// 		for (int i=0; i<magic_enum::enum_count<SceneNetwork::GradientMode>(); ++i)
// 		{
// 			SceneNetwork::GradientMode m = magic_enum::enum_value<SceneNetwork::GradientMode>(i);
// 			auto name = magic_enum::enum_name(m);
// 			bool selected = m == gradientMode_;
// 			if (ImGui::Selectable(name.data(), selected))
// 			{
// 				gradientMode_ = m;
// 				changed = true;
// 			}
// 			if (selected)
// 				ImGui::SetItemDefaultFocus();
// 		}
// 		ImGui::EndCombo();
// 	}
// 	if (gradientMode_ == SceneNetwork::GradientMode::FINITE_DIFFERENCES)
// 	{
// 		float divStepsize = 1 / finiteDifferencesStepsize_;
// 		if (ImGui::SliderFloat("1 / FD Stepsize##Network", &divStepsize, 32, 1024, "%.1f", 2))
// 		{
// 			finiteDifferencesStepsize_ = 1 / divStepsize;
// 			changed = true;
// 		}
// 	}
// 
// 	if (!networks_.empty())
// 	{
// 		auto net = networks_[selectedNetwork_].network;
// 		std::stringstream layers;
// 		for (int i = 0; i < net->numLayers(); ++i)
// 		{
// 			if (i == 0) layers << net->getHidden(i)->channelsIn;
// 			layers << ":" << net->getHidden(i)->channelsOut;
// 		}
// 		std::string layerStr = layers.str();
// 		bool isAdjoint = gradientMode_ == SceneNetwork::GradientMode::ADJOINT_METHOD;
// 		ImGui::Text("Input: direction=%d, #fourier=%d\nOutput: %s\nLayers: %s (%dB)\nWarps: %d / %d",
// 			net->input()->hasDirection ? 1 : 0,
// 			net->input()->numFourierFeatures,
// 			OutputParametrization::OutputModeNames[net->output()->outputMode].c_str(),
// 			layerStr.c_str(), net->numParameters()*2 /*sizeof(half)*/,
// 			isAdjoint ? networks_[selectedNetwork_].numWarpsMixedAdjoint : networks_[selectedNetwork_].numWarpsMixed,
// 			isAdjoint ? networks_[selectedNetwork_].numWarpsSharedOnlyAdjoint : networks_[selectedNetwork_].numWarpsSharedOnly);
// 		if (net->latentGrid())
// 		{
// 			ImGui::Text("Grid: res=%d^3, channels=%d",
// 				net->latentGrid()->getResolution(), net->latentGrid()->getTotalChannels());
// 			ImGui::Text("Time keyframes: %d, ensembles: %d",
// 				net->latentGrid()->timeNum, net->latentGrid()->ensembleNum);
// 		}
// 	}
// 
// 	return changed;
// }

// void renderer::VolumeInterpolationNetwork::load(const nlohmann::json& json, const ILoadingContext* fetcher)
// {
// 	gradientMode_ = magic_enum::enum_cast<SceneNetwork::GradientMode>(json.value("gradientMode", ""))
// 		.value_or(SceneNetwork::GradientMode::OFF_OR_DIRECT);
// 	finiteDifferencesStepsize_ = json.value("finiteDifferencesStepsize", 1 / 256.f);
// 
// 	//load the networks
// 	//TODO
// }

// void renderer::VolumeInterpolationNetwork::save(nlohmann::json& json, const ISavingContext* context) const
// {
// 	json["gradientMode"] = magic_enum::enum_name(gradientMode_);
// 	json["finiteDifferencesStepsize"] = finiteDifferencesStepsize_;
// 
// 	//save the networks
// 	//TODO
// }

int renderer::VolumeInterpolationNetwork::getCurrentNumWarps() const
{
	if (networks_.empty())
		throw std::runtime_error("No network specified!");
	const auto& net = networks_[selectedNetwork_];
	bool isAdjoint = gradientMode_ == SceneNetwork::GradientMode::ADJOINT_METHOD;
	int warpsSharedOnly = isAdjoint ? net.numWarpsSharedOnlyAdjoint : net.numWarpsSharedOnly;
	int warpsMixed = isAdjoint ? net.numWarpsMixedAdjoint : net.numWarpsMixed;
	if (warpsMixed > 0 && warpsSharedOnly > 0)
		return onlySharedMemory_ ? warpsSharedOnly : warpsMixed;
	return warpsMixed;
}

void renderer::VolumeInterpolationNetwork::prepareRendering(GlobalSettings& s) const
{
	// prepareRendering(s);
	if (networks_.empty())
		throw std::runtime_error("No network specified!");

	s.synchronizedThreads = true;
	const auto& net = networks_[selectedNetwork_];

	bool isAdjoint = gradientMode_ == SceneNetwork::GradientMode::ADJOINT_METHOD;
	int warpsSharedOnly = isAdjoint ? net.numWarpsSharedOnlyAdjoint : net.numWarpsSharedOnly;
	int warpsMixed = isAdjoint ? net.numWarpsMixedAdjoint : net.numWarpsMixed;

	if (warpsMixed > 0 && warpsSharedOnly > 0) {
		currentNumWarps_ = onlySharedMemory_ ? warpsSharedOnly : warpsMixed;
		currentOnlyShared_ = onlySharedMemory_;
	}
	else
	{
		if (warpsMixed <= 0)
			throw std::runtime_error("Network is too large!");
		currentNumWarps_ = warpsMixed;
		currentOnlyShared_ = false;
	}
	currentNumWarps_ = min(currentNumWarps_, MAX_BLOCK_SIZE / 32);
	currentTargetBlockSize_ = currentNumWarps_ * 32;

	if (s.fixedBlockSize > 0 && s.fixedBlockSize != currentTargetBlockSize_)
		throw std::runtime_error("Another module already requested a different, fixed block size");
	s.fixedBlockSize = currentTargetBlockSize_;
}

renderer::GlobalSettings::VolumeOutput renderer::VolumeInterpolationNetwork::outputType() const
{
	if (networks_.empty())
		throw std::runtime_error("No network specified!");
	const auto& net = networks_[selectedNetwork_];
	int c = net.network->output()->channelsOut();
	for (int i=0; i<3; ++i)
	{
		if (OutputType2ChannelCount[i] == c)
			return static_cast<GlobalSettings::VolumeOutput>(i);
	}
	throw std::runtime_error("Unknown number of output channels, not compatible to one of the pre-defined output types");
}

// std::optional<int> renderer::VolumeInterpolationNetwork::getBatches(const GlobalSettings& s) const
// {
// 	return {};
// }

std::string renderer::VolumeInterpolationNetwork::getDefines(const GlobalSettings& s) const
{
	if (networks_.empty())
		throw std::runtime_error("No network specified, can't render!");
	const auto& net = networks_[selectedNetwork_];
	
	if (s.volumeShouldProvideNormals) {
		switch (gradientMode_)
		{
		case SceneNetwork::GradientMode::OFF_OR_DIRECT:
			if (!net.network->supportsNormals())
				std::cerr << "WARNING: normals requested, but the SceneNetwork can't directly provide those, or FD/AD gradients are disabled" << std::endl;
			break;
		case SceneNetwork::GradientMode::FINITE_DIFFERENCES:
		case SceneNetwork::GradientMode::ADJOINT_METHOD:
			if (net.network->output()->isColor())
				std::cerr << "WARNING: normales requested, but color-predicting networks can't provide those" << std::endl;
			break;
		}
	}
	if (s.volumeShouldProvideCurvature)
	{
		if (!net.network->output()->isDensityCurvature())
		{
			throw std::runtime_error("curvature can only be provided for network output mode 'densitycurvature");
		}
		if (gradientMode_ != SceneNetwork::GradientMode::OFF_OR_DIRECT)
		{
			throw std::runtime_error("curvature can only be provided directly be the network. The Adjoint-method or finite differences is not supported yet.");
		}
	}

	return net.network->getDefines(s, currentNumWarps_, currentOnlyShared_, gradientMode_);
}

std::vector<std::string> renderer::VolumeInterpolationNetwork::getIncludeFileNames(const GlobalSettings& s) const
{
	if (networks_.empty())
		throw std::runtime_error("No network specified, can't render!");
	return networks_[selectedNetwork_].network->getIncludeFileNames(s);
}

std::string renderer::VolumeInterpolationNetwork::getConstantDeclarationName(const GlobalSettings& s) const
{
	if (networks_.empty())
		throw std::runtime_error("No network specified, can't render!");
	return networks_[selectedNetwork_].network->getConstantDeclarationName(s);
}

std::string renderer::VolumeInterpolationNetwork::getPerThreadType(const GlobalSettings& s) const
{
	if (networks_.empty())
		throw std::runtime_error("No network specified, can't render!");
	return networks_[selectedNetwork_].network->getPerThreadType(s);
}

void renderer::VolumeInterpolationNetwork::fillConstantMemory(const GlobalSettings& s, hipDeviceptr_t ptr, hipStream_t stream)
{
	if (networks_.empty()) 
		throw std::runtime_error("No network specified, can't render!");
	auto net = networks_[selectedNetwork_].network;
	float stepsize = finiteDifferencesStepsize_;
	if (gradientMode_ == SceneNetwork::GradientMode::ADJOINT_METHOD) {
		int gridResolution = net->latentGrid() ? net->latentGrid()->getResolution() : 1;
		stepsize = 1 / (static_cast<float>(gridResolution) * adjointLatentGridCentralDifferencesStepsizeScale_);
	}
	net->fillConstantMemory(
		s, stepsize, ptr, stream);
}
